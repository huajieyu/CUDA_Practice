#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void global_reduce_kernel(float *d_out, float *d_in, int n)
{
    int myId = threadIdx + blockIdx.x * blockDim.x; //global Id
    int tid = threadIdx.x; //threadId

    //do reduction in global memory
    for(unsigned int s= blockDim.x/2; s>0; s>>=1)
    {
        if(tid<2)
        {
            d_in[myId] += d_in[myId+x];
        }
        __syncthreads();  //make sure all adds at one stage are done!
    }

    //only thread 0  writes results for this block back to global memory
    if(tid ==0)
    {
        d_out[blockIdx.x] = d_in[myId];
    }
}

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 1024; // Number of elements
    const int BLOCK_SIZE = 256; // Number of threads per block
    const int NUM_BLOCKS = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float h_in[N], h_out[NUM_BLOCKS];

    // Initialize input data
    for (int i = 0; i < N; i++) {
        h_in[i] = 1.0f; // Example: set all values to 1.0
    }

    float *d_in, *d_out;
    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, NUM_BLOCKS * sizeof(float));
    
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    global_reduce_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_out, d_in, N);
    checkCUDAError("Kernel launch failed");

    // Synchronize and record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Copy the result back to the host
    hipMemcpy(h_out, d_out, NUM_BLOCKS * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate total reduction
    float total = 0;
    for (int i = 0; i < NUM_BLOCKS; i++) {
        total += h_out[i];
    }

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Output results
    printf("Total: %f\n", total);
    printf("Execution time: %f ms\n", milliseconds);

    // Cleanup
    hipFree(d_in);
    hipFree(d_out);
    
    return 0;
}